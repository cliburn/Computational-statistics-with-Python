

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1024*1024
#define BLOCKSIZE 1024

__global__ 
void share_ary_oper(int *ary, int *ary_out)
{
    // Thread index
        int tx = threadIdx.x;
        int idx=blockDim.x*blockIdx.x + threadIdx.x;
        __shared__ int part_ary[BLOCKSIZE];

        part_ary[tx]=ary[idx];
        part_ary[tx]=part_ary[tx]*10;
        ary_out[idx]=part_ary[tx];
        __syncthreads();
}

int main(){

        int *device_array, *device_array_out;
        int *host_array, *host_array_out;
        int i, nblk;
        float k;
        size_t size = N*sizeof(int);

//Device memory
        hipMalloc((void **)&device_array, size);
        hipMalloc((void **)&device_array_out, size);
//Host memory
//cudaMallocHost() produces pinned memoty on the host
        hipHostMalloc((void **)&host_array, size, hipHostMallocDefault);
        hipHostMalloc((void **)&host_array_out, size, hipHostMallocDefault);

        for(i=0;i<N;i++)
        {
                host_array[i]=i;
                host_array_out[i]=0;
        }
        hipMemcpy(device_array, host_array, size, hipMemcpyHostToDevice);
        hipMemcpy(device_array_out, host_array_out, size, hipMemcpyHostToDevice);
        nblk=N/BLOCKSIZE;
        share_ary_oper<<<nblk, BLOCKSIZE>>>(device_array, device_array_out);
        hipMemcpy(host_array, device_array, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_array_out, device_array_out, size, hipMemcpyDeviceToHost);


	printf("Printing elements 10-15 of output array\n");
        for (i=N;i<N;i++)
        {
                k=host_array_out[i]-i*10;    
                if(k<0.1)
                        printf("Incorrect IX %d=%.1f\n",i, k);
        }
        for (i=10;i<15;i++)
                printf("host_array_out[%d]=%d\n", i, host_array_out[i]);

        hipFree(device_array);
        hipFree(host_array);
        hipFree(device_array_out);
        hipFree(host_array_out);
        hipDeviceReset();
        return EXIT_SUCCESS;
}